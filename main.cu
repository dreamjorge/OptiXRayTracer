#include <optix.h>
#include <iostream>
#include <cstring>

// Error checking macro for CUDA
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl; \
            return 1; \
        } \
    } while (0)

int main() {
    // Initialize OptiX library
    if (optixInit() != OPTIX_SUCCESS) {
        std::cerr << "Failed to initialize OptiX." << std::endl;
        return 1;
    }

    // Set up device context
    hipCtx_t cuCtx = 0;
    OptixDeviceContext optixContext;
    OptixDeviceContextOptions options = {};
    options.logCallbackFunction = nullptr;
    options.logCallbackLevel = 4;

    if (optixDeviceContextCreate(cuCtx, &options, &optixContext) != OPTIX_SUCCESS) {
        std::cerr << "Failed to create OptiX context." << std::endl;
        return 1;
    }

    // Load PTX code for ray generation shader
    const char* ptxCode = R"(
        .version 7.0
        .target sm_70
        .address_size 64

        // Ray generation program
        .entry __raygen__simple {
            // For simplicity, empty shader
        }
    )";

    // Compile and load module
    OptixModule module;
    OptixModuleCompileOptions module_compile_options = {};
    OptixPipelineCompileOptions pipeline_compile_options = {};
    pipeline_compile_options.usesMotionBlur = false;
    pipeline_compile_options.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;

    if (optixModuleCreateFromPTX(optixContext, &module_compile_options, &pipeline_compile_options,
                                 ptxCode, strlen(ptxCode), nullptr, 0, &module) != OPTIX_SUCCESS) {
        std::cerr << "Failed to create module." << std::endl;
        return 1;
    }

    // Create ray generation program group
    OptixProgramGroup raygen_prog_group;
    OptixProgramGroupOptions program_group_options = {};
    OptixProgramGroupDesc raygen_prog_group_desc = {};
    raygen_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
    raygen_prog_group_desc.raygen.module = module;
    raygen_prog_group_desc.raygen.entryFunctionName = "__raygen__simple";

    if (optixProgramGroupCreate(optixContext, &raygen_prog_group_desc, 1, &program_group_options,
                                nullptr, nullptr, &raygen_prog_group) != OPTIX_SUCCESS) {
        std::cerr << "Failed to create program group." << std::endl;
        return 1;
    }

    // Pipeline creation
    OptixPipeline pipeline;
    OptixPipelineLinkOptions pipeline_link_options = {};
    pipeline_link_options.maxTraceDepth = 1;
    
    if (optixPipelineCreate(optixContext, &pipeline_compile_options, &pipeline_link_options,
                            &raygen_prog_group, 1, nullptr, nullptr, &pipeline) != OPTIX_SUCCESS) {
        std::cerr << "Failed to create pipeline." << std::endl;
        return 1;
    }

    std::cout << "OptiX pipeline created successfully!" << std::endl;

    // Clean up
    optixPipelineDestroy(pipeline);
    optixProgramGroupDestroy(raygen_prog_group);
    optixModuleDestroy(module);
    optixDeviceContextDestroy(optixContext);

    return 0;
}
